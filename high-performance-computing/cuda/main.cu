
#include <hip/hip_runtime.h>
#include <cstdio>

#define ARR_SIZE 10

void __global__ sum_array(int* d_arr, int* res, size_t param){
  printf("inside sum_array\n");
  printf("param: %lu\n", param);
  for(int i = 0; i < ARR_SIZE; i++){
    *res += d_arr[i];
  }
}

int main(){
  int arr[ARR_SIZE] = {1, 0, 3, 5, 0, 0, 0, 6, 7, 10};
  int expected_sum = 32;
  int actual_sum = 0;
  size_t number = 13;

  int *d_sum, *d_arr;

  hipMalloc(&d_sum, sizeof(int));
  hipMalloc(&d_arr, sizeof(int)*ARR_SIZE);
  hipMemcpy(d_sum, &actual_sum, sizeof(int), hipMemcpyDefault);
  hipMemcpy(d_arr, &arr, sizeof(int)*ARR_SIZE, hipMemcpyDefault);
  sum_array<<<1, 1>>>(d_arr, d_sum, number);
  hipDeviceSynchronize();
  hipMemcpy(&actual_sum, d_sum, sizeof(int), hipMemcpyDefault);

  printf("actual_sum: %i\n", actual_sum);
  printf("expected_sum: %i\n", expected_sum);
}

