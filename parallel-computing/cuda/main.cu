
#include <hip/hip_runtime.h>
#include <cstdio>

#define ARR_SIZE 10

void __global__ sum_array(int* d_arr, int* res){
  printf("inside sum_array\n");
  for(int i = 0; i < ARR_SIZE; i++){
    res += d_arr[i];
  }
}

int main(){
  int arr[ARR_SIZE] = {1, 0, 3, 5, 0, 0, 0, 6, 7, 10};
  int expected_sum = 32;
  int actual_sum = 0;

  int *d_sum, *d_arr;

  hipMalloc(&d_sum, sizeof(int));
  hipMalloc(&d_arr, sizeof(int)*ARR_SIZE);
  hipMemcpy(d_sum, &actual_sum, sizeof(int), hipMemcpyDefault);
  hipMemcpy(d_arr, &arr, sizeof(int)*ARR_SIZE, hipMemcpyDefault);
  sum_array<<<1, 1>>>(d_arr, d_sum);
  hipDeviceSynchronize();
  hipMemcpy(&actual_sum, d_sum, sizeof(int), hipMemcpyDefault);

  printf("actual_sum: %i\n", actual_sum);
  printf("expected_sum: %i\n", expected_sum);
}

